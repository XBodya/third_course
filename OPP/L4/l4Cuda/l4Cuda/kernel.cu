#include "hip/hip_runtime.h"
﻿#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "time.h"

#define ifStatusIsFail if(cudaStatus != hipSuccess) 
#define check cudaStatus =
#define INIT_TIMER time_t start; double __time_in_sec;
#define START_TIMER start = time(0)
#define STOP_TIMER __time_in_sec = difftime(time(0), start);
#define PRINT_TIME(message) std::cout << message << " @Time execution: " << __time_in_sec << '\n';


__global__ void transposeKernel(float* out, const float* in, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < height && j < width) {
        out[j * height + i] = in[i * width + j];
    }
}

void transpose(float* arr, float* out_arr, int width, int height) {
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            out_arr[j * height + i] = arr[i * width + j];
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Time to exec by cpu:  %3.1f ms \n", time);
}

void initMatrix(float* matrix, int width, int height, int c=1) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            matrix[i * width + j] = (i * width + j)*c;
        }
    }
}

void print(const float* matrix, int width, int height) {
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
}

hipError_t cudaTranspose(float *host_in, float *host_out, int width, int height) {
    size_t size = width * height * sizeof(float);
    hipError_t cudaStatus = hipSuccess;
    float* device_in;
    float* device_out;

    float time;
    hipEvent_t start, stop;

    check hipMalloc(&device_in, size);
    ifStatusIsFail{
        std::cout << "device_in malloc fail\n";
        return cudaStatus;
    }
    check hipMalloc(&device_out, size);
    ifStatusIsFail{
        std::cout << "device_out malloc fail\n";
        return cudaStatus;
    }

    check hipMemcpy(device_in, host_in, size, hipMemcpyHostToDevice);
    ifStatusIsFail{
        std::cout << "device_in copy fail\n";
        return cudaStatus;
    }
    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (width + threadsPerBlock.y - 1) / threadsPerBlock.y);
    //START_TIMER;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    transposeKernel << <numBlocks, threadsPerBlock >> > (device_out, device_in, width, height);
    //STOP_TIMER;
    //PRINT_TIME("Kernel time exec: ");

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Time to exec cuda gpu kernel:  %3.1f ms \n", time);

    cudaStatus = hipGetLastError();
    ifStatusIsFail{
        std::cout << "transpose fail\n";
        return cudaStatus;
    }

    check hipMemcpy(host_out, device_out, size, hipMemcpyDeviceToHost);
    ifStatusIsFail{
        std::cout << "copy device2host fail\n";
        return cudaStatus;
    }

    hipFree(device_in);
    hipFree(device_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return cudaStatus;
}

int main() {
    INIT_TIMER;
    int w = 60000, h = 20000;
    size_t size = w * h * sizeof(float);
    float* h_in = (float*)malloc(size);
    float* host_out = (float*)malloc(size);
    initMatrix(h_in, w, h);
    START_TIMER;
    cudaTranspose(h_in, host_out, w, h);
    //print(h_in, w, h);
    //std::cout << '\n';
    //print(host_out, h, w);
    STOP_TIMER;
    //PRINT_TIME("Cuda trans.");
    START_TIMER;
    initMatrix(h_in, w, h, 2);
    transpose(h_in, host_out, w, h);
    //print(h_in, w, h);
    //print(host_out, h, w);
    STOP_TIMER;
    //PRINT_TIME("CPU trans.");
    free(h_in);
    free(host_out);
    return 0;
}